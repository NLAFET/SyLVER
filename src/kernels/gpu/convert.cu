#include "hip/hip_runtime.h"
/// @file
/// @copyright 2016- The Science and Technology Facilities Council (STFC)
/// @author Florent Lopez
#include "kernels/gpu/convert.cuh"
//STD
#include <cassert>
#include <iostream>
// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// MAGMA see magmablas/hlaconvert.cu
const int max_blocks = 65535;

// MAGMA see magmablas/hlaconvert.cu
#define BLK_X 64
#define BLK_Y BLK_X

namespace sylver {
namespace gpu {

   // MAGMA routine see magmablas/hlaconvert.cu
   static __device__
   void convert_sp2hp_device(
         int m, int n,
         const float  *dA, int ldda,
         sylver::gpu::half *dB, int lddb )
   {
      int ind = blockIdx.x*BLK_X + threadIdx.x;
      int iby = blockIdx.y*BLK_Y;
      /* check if full block-column */
      bool full = (iby + BLK_Y <= n);
      /* do only rows inside matrix */
      if ( ind < m ) {
         dA += ind + iby*ldda;
         dB += ind + iby*lddb;
         if ( full ) {
            // full block-column
#pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
               dB[j*lddb] = __float2half( dA[j*ldda] );
            }
         }
         else {
            // partial block-column
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
               dB[j*lddb] = __float2half( dA[j*ldda] );
            }
         }
      }
   }

   // MAGMA routine see magmablas/hlaconvert.cu
   static __device__
   void convert_hp2sp_device(
         int m, int n,
         const sylver::gpu::half *dA, int ldda,
         float  *dB, int lddb )
   {
      int ind = blockIdx.x*BLK_X + threadIdx.x;
      int iby = blockIdx.y*BLK_Y;
      /* check if full block-column */
      bool full = (iby + BLK_Y <= n);
      /* do only rows inside matrix */
      if ( ind < m ) {
         dA += ind + iby*ldda;
         dB += ind + iby*lddb;
         if ( full ) {
            // full block-column
#pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
               dB[j*lddb] = __half2float( dA[j*ldda] );
            }
         }
         else {
            // partial block-column
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
               dB[j*lddb] = __half2float( dA[j*ldda] );
            }
         }
      }
   }

   // MAGMA routine see magmablas/hlaconvert.cu
   __global__
   void convert_sp2hp_kernel(
         int m, int n,
         const float  *dA, int ldda,
         sylver::gpu::half *dB, int lddb )
   {
#if CUDA_VERSION >= 7500
      convert_sp2hp_device(m, n, dA, ldda, dB, lddb);
#endif
   }

   // MAGMA routine see magmablas/hlaconvert.cu
   __global__
   void convert_hp2sp_kernel(
         int m, int n,
         const sylver::gpu::half *dA, int ldda,
         float  *dB, int lddb )
   {
#if CUDA_VERSION >= 7500
      convert_hp2sp_device(m, n, dA, ldda, dB, lddb);
#endif
   }

   template<typename TA, typename TAO> 
   __global__
   void convert_kernel(int m, int n, const TA  *dA, int ldda, TAO *dB, int lddb );

   // Template specialization
   template<>
   __global__
   void convert_kernel<float, sylver::gpu::half>(
         int m, int n,
         const float  *dA, int ldda,
         sylver::gpu::half *dB, int lddb ) {
      convert_sp2hp_device(m, n, dA, ldda, dB, lddb);
   }
   template<>
   __global__
   void convert_kernel<sylver::gpu::half, float>(
         int m, int n,
         const sylver::gpu::half  *dA, int ldda,
         float *dB, int lddb ) {
      convert_hp2sp_device(m, n, dA, ldda, dB, lddb);
   }

   // @brief Convert matrix a of type float into half prec and put
   // result in aout
   template<typename TA, typename TAO> 
   void convert(
         hipStream_t const stream,
         int m, int n,
         TA *const a, int lda, 
         TAO *const aout, int ldaout) {
      
      std::cout << "[convert]"
                << " m = " << m << ", n = " << n
                << " lda = " << lda << ", ldaout = " << ldaout
                << std::endl;
      
      assert( BLK_X == BLK_Y );
      const int super_NB = max_blocks*BLK_X;
      dim3 super_grid(
            (m + super_NB - 1) / super_NB, 
            (n + super_NB - 1) / super_NB);
    
      dim3 threads( BLK_X, 1 );
      dim3 grid;
    
      int mm, nn;
      for( unsigned int i=0; i < super_grid.x; ++i ) {
         mm = (i == super_grid.x-1 ? m % super_NB : super_NB);
         grid.x = (mm + super_NB - 1) / BLK_X;
         for( unsigned int j=0; j < super_grid.y; ++j ) {  // full row
            nn = (j == super_grid.y-1 ? n % super_NB : super_NB);
            grid.y = (nn + super_NB - 1) / BLK_Y;
            convert_kernel 
               <<< grid, threads, 0, stream >>>
               (mm, nn, &a[i*super_NB + j*super_NB*lda], lda, &aout[i*super_NB + j*super_NB*ldaout], ldaout);
         }
      }
   }
   
   template void convert<float, sylver::gpu::half>(
         hipStream_t const stream, int m, int n, float *const a, int lda, 
         sylver::gpu::half *const aout, int ldaout);

   template void convert<sylver::gpu::half, float>(
         hipStream_t const stream, int m, int n, sylver::gpu::half *const a, int lda, 
         float *const aout, int ldaout);

}} // End of namespace sykver::gpu
