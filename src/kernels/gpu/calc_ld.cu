#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define CALC_LD_NTX 8  // Number of threads x

namespace /* anon */ {

   __global__ void
   cu_calc_ld(
         int m,
         int n,
         double *const d) {

   }

}

namespace spldlt {
   
   extern "C" {
      
      void calc_ld(
            const hipStream_t stream,
            int m,
            int n,
            double *const d,
            ) {

      }
      
   }

}
