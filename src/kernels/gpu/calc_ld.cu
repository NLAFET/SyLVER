/// @file
/// @copyright 2016- The Science and Technology Facilities Council (STFC)
/// @author Florent Lopez

#include <stdio.h>
#include <limits>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// #define BLOCK_SIZE 128  // Number of threads
// #define BLOCK_SIZE 16  // Number of threads
#define BLOCK_SIZE 32  // Number of threads

namespace /* anon */ {

   template<typename T>
   __global__ void
   cu_calc_ld(
         int m, int n,
         T *const l, int ldl,
         T *const d,
         double *ld, int ldld) {

      int bx = blockIdx.x;
      int by = blockIdx.y;

      int tx = threadIdx.x;
      int ty = threadIdx.y;

      // if (tx == 0 && ty == 0)
      //    printf("[cu_calc_ld] m = %d, n = %d\n", m, n);

      for (int col = ty + by * blockDim.y; col < n; col += blockDim.y * gridDim.y) {
         
         // Check if we are halfway trhough a 2x2 pivot
         // if (col+1 < n && !std::isfinite(d[2*col]))
         //    continue;
         
         if((col+1==n || std::isfinite(d[2*col+2])) && std::isfinite(d[2*col])) {
            // 1x1 pivot

            T d11 = d[2*col];
            if(d11 != 0.0) d11 = 1/d11;

            for(int row = tx + bx * blockDim.x; row < m; row += blockDim.x * gridDim.x)
               ld[col*ldld+row] = d11 * l[col*ldl+row];

         }
         // else {
         else if (std::isfinite(d[2*col])) {
         // else if (d[2*col]==std::numeric_limits<T>::infinity()) {
            // 2x2 pivot

            T d11 = d[2*col];
            T d21 = d[2*col+1];
            T d22 = d[2*col+3];

            // printf("[cu_calc_ld] d11 = %f, d21 = %f, d22 = %f\n", d11, d21, d22);

            T det = d11*d22 - d21*d21;
            d11 = d11/det;
            d21 = d21/det;
            d22 = d22/det;
            for(int row = tx + bx * blockDim.x; row < m; row += blockDim.x * gridDim.x) {
               T a1, a2;
               a1 = l[col*ldl+row];
               a2 = l[(col+1)*ldl+row];
               ld[col*ldld+row]     =  d22*a1 - d21*a2;
               ld[(col+1)*ldld+row] = -d21*a1 + d11*a2;
            }

         }
      }

   }

   // template<typename T>
   // __global__ void
   // cu_calc_ld(
   //       int m,
   //       int n,
   //       T *const l, int ldl,
   //       T *const d,
   //       double *ld, int ldld) {

   //    // printf("[cu_calc_ld] blockIdx.x = %d\n", blockIdx.x);
      
   //    for (int col = 0; col < n; ) {

   //       if(col+1==n || std::isfinite(d[2*col+2])) {
   //          // 1x1 pivot

   //          // printf("[cu_calc_ld] 1x1\n");

   //          T d11 = d[2*col];
   //          if(d11 != 0.0) d11 = 1/d11;

   //          for(int row = threadIdx.x + blockIdx.x * blockDim.x; row < m; row += blockDim.x * gridDim.x)
   //             ld[col*ldld+row] = d11 * l[col*ldl+row];

   //          col++;
   //       }
   //       else {
   //          // 2x2 pivot

   //          // printf("[cu_calc_ld] 2x2\n");

   //          T d11 = d[2*col];
   //          T d21 = d[2*col+1];
   //          T d22 = d[2*col+3];

   //          // printf("[cu_calc_ld] d11 = %f, d21 = %f, d22 = %f\n", d11, d21, d22);

   //          T det = d11*d22 - d21*d21;
   //          d11 = d11/det;
   //          d21 = d21/det;
   //          d22 = d22/det;
   //          for(int row = threadIdx.x + blockIdx.x * blockDim.x; row < m; row += blockDim.x * gridDim.x) {
   //             T a1, a2;
   //             a1 = l[col*ldl+row];
   //             a2 = l[(col+1)*ldl+row];
   //             ld[col*ldld+row]     =  d22*a1 - d21*a2;
   //             ld[(col+1)*ldld+row] = -d21*a1 + d11*a2;
   //          }
            
   //          col += 2;
   //       }
   //    }
   // }

} // end of anon namespace

namespace sylver {
namespace spldlt {
namespace gpu {
   
   // template<typename T>
   void calc_ld(
         const hipStream_t stream,
         int m, int n,
         double *const l, int ldl,
         double *const d,
         double *ld, int ldld
         ) {

      dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
      dim3 grid((m + threads.x - 1) / threads.x, (n + threads.y -1) / threads.y);

      // printf("[calc_ld] m = %d, n = %d\n", m, n);
      // printf("[calc_ld] thx = %d, thy = %d\n", threads.x, threads.y);
      // printf("[calc_ld] grx = %d, gry = %d\n", grid.x, grid.y);

      // int nb = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
         
      cu_calc_ld
         <double>
         // <<<1, 1, 0, stream>>>
         // <<<nb, BLOCK_SIZE, 0, stream>>>
         <<<grid, threads, 0, stream>>>
         (m, n, l, ldl, d, ld, ldld);

   }
}}} // End of namespace sylver::spldlt::gpu
