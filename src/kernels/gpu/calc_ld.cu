#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 8  // Number of threads

namespace /* anon */ {

   __global__ void
   cu_calc_ld(
         int m,
         int n,
         double *const d) {

   }

}

namespace spldlt {
   
   extern "C" {
      
      void calc_ld(
            const hipStream_t stream,
            int m,
            int n,
            double *const d
            ) {
         
      }
      
   }

}
